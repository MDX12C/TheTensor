#include "hip/hip_runtime.h"
#include "thetensor/define.h"
#include "thetensor/parallel.cuh"

#ifdef __cplusplus
extern "C"{
#endif
//#####################################################################################################################
__global__ void __vv_add_b(bool* __a, bool* __b, bool* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a[i] | __b[i];
  return;
}
/*the add of bool in vector to vector*/
void vv_add_b(bool* __a, bool* __b, bool* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __vv_add_b<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __vv_add_i8(int8_t* __a, int8_t* __b, int8_t* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a[i] + __b[i];
  return;
}
/*the add of int8_t in vector to vector*/
void vv_add_i8(int8_t* __a, int8_t* __b, int8_t* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __vv_add_i8<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __vv_add_i16(int16_t* __a, int16_t* __b, int16_t* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a[i] + __b[i];
  return;
}
/*the add of int16_t in vector to vector*/
void vv_add_i16(int16_t* __a, int16_t* __b, int16_t* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __vv_add_i16<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __vv_add_i32(int32_t* __a, int32_t* __b, int32_t* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a[i] + __b[i];
  return;
}
/*the add of int32_t in vector to vector*/
void vv_add_i32(int32_t* __a, int32_t* __b, int32_t* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __vv_add_i32<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __vv_add_i64(int64_t* __a, int64_t* __b, int64_t* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a[i] + __b[i];
  return;
}
/*the add of int64_t in vector to vector*/
void vv_add_i64(int64_t* __a, int64_t* __b, int64_t* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __vv_add_i64<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __vv_add_u8(uint8_t* __a, uint8_t* __b, uint8_t* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a[i] + __b[i];
  return;
}
/*the add of uint8_t in vector to vector*/
void vv_add_u8(uint8_t* __a, uint8_t* __b, uint8_t* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __vv_add_u8<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __vv_add_u16(uint16_t* __a, uint16_t* __b, uint16_t* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a[i] + __b[i];
  return;
}
/*the add of uint16_t in vector to vector*/
void vv_add_u16(uint16_t* __a, uint16_t* __b, uint16_t* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __vv_add_u16<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __vv_add_u32(uint32_t* __a, uint32_t* __b, uint32_t* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a[i] + __b[i];
  return;
}
/*the add of uint32_t in vector to vector*/
void vv_add_u32(uint32_t* __a, uint32_t* __b, uint32_t* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __vv_add_u32<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __vv_add_u64(uint64_t* __a, uint64_t* __b, uint64_t* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a[i] + __b[i];
  return;
}
/*the add of uint64_t in vector to vector*/
void vv_add_u64(uint64_t* __a, uint64_t* __b, uint64_t* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __vv_add_u64<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __vv_add_f(float_t* __a, float_t* __b, float_t* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a[i] + __b[i];
  return;
}
/*the add of float_t in vector to vector*/
void vv_add_f(float_t* __a, float_t* __b, float_t* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __vv_add_f<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __vv_add_d(double_t* __a, double_t* __b, double_t* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a[i] + __b[i];
  return;
}
/*the add of double_t in vector to vector*/
void vv_add_d(double_t* __a, double_t* __b, double_t* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __vv_add_d<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}
//#####################################################################################################################
__global__ void __vs_add_b(bool* __a, bool __b, bool* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a[i] | __b;
  return;
}
/*the add of bool in vector to scalar*/
void vs_add_b(bool* __a, bool __b, bool* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __vs_add_b<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __vs_add_i8(int8_t* __a, int8_t __b, int8_t* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a[i] + __b;
  return;
}
/*the add of int8_t in vector to scalar*/
void vs_add_i8(int8_t* __a, int8_t __b, int8_t* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __vs_add_i8<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __vs_add_i16(int16_t* __a, int16_t __b, int16_t* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a[i] + __b;
  return;
}
/*the add of int16_t in vector to scalar*/
void vs_add_i16(int16_t* __a, int16_t __b, int16_t* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __vs_add_i16<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __vs_add_i32(int32_t* __a, int32_t __b, int32_t* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a[i] + __b;
  return;
}
/*the add of int32_t in vector to scalar*/
void vs_add_i32(int32_t* __a, int32_t __b, int32_t* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __vs_add_i32<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __vs_add_i64(int64_t* __a, int64_t __b, int64_t* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a[i] + __b;
  return;
}
/*the add of int64_t in vector to scalar*/
void vs_add_i64(int64_t* __a, int64_t __b, int64_t* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __vs_add_i64<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __vs_add_u8(uint8_t* __a, uint8_t __b, uint8_t* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a[i] + __b;
  return;
}
/*the add of uint8_t in vector to scalar*/
void vs_add_u8(uint8_t* __a, uint8_t __b, uint8_t* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __vs_add_u8<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __vs_add_u16(uint16_t* __a, uint16_t __b, uint16_t* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a[i] + __b;
  return;
}
/*the add of uint16_t in vector to scalar*/
void vs_add_u16(uint16_t* __a, uint16_t __b, uint16_t* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __vs_add_u16<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __vs_add_u32(uint32_t* __a, uint32_t __b, uint32_t* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a[i] + __b;
  return;
}
/*the add of uint32_t in vector to scalar*/
void vs_add_u32(uint32_t* __a, uint32_t __b, uint32_t* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __vs_add_u32<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __vs_add_u64(uint64_t* __a, uint64_t __b, uint64_t* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a[i] + __b;
  return;
}
/*the add of uint64_t in vector to scalar*/
void vs_add_u64(uint64_t* __a, uint64_t __b, uint64_t* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __vs_add_u64<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __vs_add_f(float_t* __a, float_t __b, float_t* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a[i] + __b;
  return;
}
/*the add of float_t in vector to scalar*/
void vs_add_f(float_t* __a, float_t __b, float_t* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __vs_add_f<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __vs_add_d(double_t* __a, double_t __b, double_t* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a[i] + __b;
  return;
}
/*the add of double_t in vector to scalar*/
void vs_add_d(double_t* __a, double_t __b, double_t* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __vs_add_d<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}
//#####################################################################################################################
__global__ void __vv_mns_b(bool* __a, bool* __b, bool* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a[i] & (~__b[i]);
  return;
}
/*the minus of bool in vector to vector*/
void vv_mns_b(bool* __a, bool* __b, bool* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __vv_mns_b<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __vv_mns_i8(int8_t* __a, int8_t* __b, int8_t* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a[i] - __b[i];
  return;
}
/*the minus of int8_t in vector to vector*/
void vv_mns_i8(int8_t* __a, int8_t* __b, int8_t* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __vv_mns_i8<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __vv_mns_i16(int16_t* __a, int16_t* __b, int16_t* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a[i] - __b[i];
  return;
}
/*the minus of int16_t in vector to vector*/
void vv_mns_i16(int16_t* __a, int16_t* __b, int16_t* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __vv_mns_i16<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __vv_mns_i32(int32_t* __a, int32_t* __b, int32_t* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a[i] - __b[i];
  return;
}
/*the minus of int32_t in vector to vector*/
void vv_mns_i32(int32_t* __a, int32_t* __b, int32_t* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __vv_mns_i32<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __vv_mns_i64(int64_t* __a, int64_t* __b, int64_t* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a[i] - __b[i];
  return;
}
/*the minus of int64_t in vector to vector*/
void vv_mns_i64(int64_t* __a, int64_t* __b, int64_t* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __vv_mns_i64<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __vv_mns_u8(uint8_t* __a, uint8_t* __b, uint8_t* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a[i] - __b[i];
  return;
}
/*the minus of uint8_t in vector to vector*/
void vv_mns_u8(uint8_t* __a, uint8_t* __b, uint8_t* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __vv_mns_u8<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __vv_mns_u16(uint16_t* __a, uint16_t* __b, uint16_t* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a[i] - __b[i];
  return;
}
/*the minus of uint16_t in vector to vector*/
void vv_mns_u16(uint16_t* __a, uint16_t* __b, uint16_t* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __vv_mns_u16<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __vv_mns_u32(uint32_t* __a, uint32_t* __b, uint32_t* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a[i] - __b[i];
  return;
}
/*the minus of uint32_t in vector to vector*/
void vv_mns_u32(uint32_t* __a, uint32_t* __b, uint32_t* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __vv_mns_u32<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __vv_mns_u64(uint64_t* __a, uint64_t* __b, uint64_t* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a[i] - __b[i];
  return;
}
/*the minus of uint64_t in vector to vector*/
void vv_mns_u64(uint64_t* __a, uint64_t* __b, uint64_t* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __vv_mns_u64<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __vv_mns_f(float_t* __a, float_t* __b, float_t* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a[i] - __b[i];
  return;
}
/*the minus of float_t in vector to vector*/
void vv_mns_f(float_t* __a, float_t* __b, float_t* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __vv_mns_f<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __vv_mns_d(double_t* __a, double_t* __b, double_t* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a[i] - __b[i];
  return;
}
/*the minus of double_t in vector to vector*/
void vv_mns_d(double_t* __a, double_t* __b, double_t* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __vv_mns_d<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}
//#####################################################################################################################
__global__ void __vs_mns_b(bool* __a, bool __b, bool* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a[i] & (~__b);
  return;
}
/*the minus of bool in vector to scalar*/
void vs_mns_b(bool* __a, bool __b, bool* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __vs_mns_b<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __vs_mns_i8(int8_t* __a, int8_t __b, int8_t* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a[i] - __b;
  return;
}
/*the minus of int8_t in vector to scalar*/
void vs_mns_i8(int8_t* __a, int8_t __b, int8_t* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __vs_mns_i8<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __vs_mns_i16(int16_t* __a, int16_t __b, int16_t* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a[i] - __b;
  return;
}
/*the minus of int16_t in vector to scalar*/
void vs_mns_i16(int16_t* __a, int16_t __b, int16_t* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __vs_mns_i16<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __vs_mns_i32(int32_t* __a, int32_t __b, int32_t* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a[i] - __b;
  return;
}
/*the minus of int32_t in vector to scalar*/
void vs_mns_i32(int32_t* __a, int32_t __b, int32_t* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __vs_mns_i32<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __vs_mns_i64(int64_t* __a, int64_t __b, int64_t* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a[i] - __b;
  return;
}
/*the minus of int64_t in vector to scalar*/
void vs_mns_i64(int64_t* __a, int64_t __b, int64_t* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __vs_mns_i64<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __vs_mns_u8(uint8_t* __a, uint8_t __b, uint8_t* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a[i] - __b;
  return;
}
/*the minus of uint8_t in vector to scalar*/
void vs_mns_u8(uint8_t* __a, uint8_t __b, uint8_t* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __vs_mns_u8<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __vs_mns_u16(uint16_t* __a, uint16_t __b, uint16_t* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a[i] - __b;
  return;
}
/*the minus of uint16_t in vector to scalar*/
void vs_mns_u16(uint16_t* __a, uint16_t __b, uint16_t* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __vs_mns_u16<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __vs_mns_u32(uint32_t* __a, uint32_t __b, uint32_t* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a[i] - __b;
  return;
}
/*the minus of uint32_t in vector to scalar*/
void vs_mns_u32(uint32_t* __a, uint32_t __b, uint32_t* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __vs_mns_u32<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __vs_mns_u64(uint64_t* __a, uint64_t __b, uint64_t* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a[i] - __b;
  return;
}
/*the minus of uint64_t in vector to scalar*/
void vs_mns_u64(uint64_t* __a, uint64_t __b, uint64_t* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __vs_mns_u64<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __vs_mns_f(float_t* __a, float_t __b, float_t* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a[i] - __b;
  return;
}
/*the minus of float_t in vector to scalar*/
void vs_mns_f(float_t* __a, float_t __b, float_t* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __vs_mns_f<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __vs_mns_d(double_t* __a, double_t __b, double_t* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a[i] - __b;
  return;
}
/*the minus of double_t in vector to scalar*/
void vs_mns_d(double_t* __a, double_t __b, double_t* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __vs_mns_d<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}
//#####################################################################################################################
__global__ void __sv_mns_b(bool __a, bool* __b, bool* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a & (~__b[i]);
  return;
}
/*the minus of bool in scalar to vector*/
void sv_mns_b(bool __a, bool* __b, bool* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __sv_mns_b<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __sv_mns_i8(int8_t __a, int8_t* __b, int8_t* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a - __b[i];
  return;
}
/*the minus of int8_t in scalar to vector*/
void sv_mns_i8(int8_t __a, int8_t* __b, int8_t* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __sv_mns_i8<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __sv_mns_i16(int16_t __a, int16_t* __b, int16_t* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a - __b[i];
  return;
}
/*the minus of int16_t in scalar to vector*/
void sv_mns_i16(int16_t __a, int16_t* __b, int16_t* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __sv_mns_i16<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __sv_mns_i32(int32_t __a, int32_t* __b, int32_t* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a - __b[i];
  return;
}
/*the minus of int32_t in scalar to vector*/
void sv_mns_i32(int32_t __a, int32_t* __b, int32_t* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __sv_mns_i32<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __sv_mns_i64(int64_t __a, int64_t* __b, int64_t* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a - __b[i];
  return;
}
/*the minus of int64_t in scalar to vector*/
void sv_mns_i64(int64_t __a, int64_t* __b, int64_t* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __sv_mns_i64<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __sv_mns_u8(uint8_t __a, uint8_t* __b, uint8_t* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a - __b[i];
  return;
}
/*the minus of uint8_t in scalar to vector*/
void sv_mns_u8(uint8_t __a, uint8_t* __b, uint8_t* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __sv_mns_u8<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __sv_mns_u16(uint16_t __a, uint16_t* __b, uint16_t* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a - __b[i];
  return;
}
/*the minus of uint16_t in scalar to vector*/
void sv_mns_u16(uint16_t __a, uint16_t* __b, uint16_t* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __sv_mns_u16<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __sv_mns_u32(uint32_t __a, uint32_t* __b, uint32_t* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a - __b[i];
  return;
}
/*the minus of uint32_t in scalar to vector*/
void sv_mns_u32(uint32_t __a, uint32_t* __b, uint32_t* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __sv_mns_u32<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __sv_mns_u64(uint64_t __a, uint64_t* __b, uint64_t* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a - __b[i];
  return;
}
/*the minus of uint64_t in scalar to vector*/
void sv_mns_u64(uint64_t __a, uint64_t* __b, uint64_t* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __sv_mns_u64<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __sv_mns_f(float_t __a, float_t* __b, float_t* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a - __b[i];
  return;
}
/*the minus of float_t in scalar to vector*/
void sv_mns_f(float_t __a, float_t* __b, float_t* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __sv_mns_f<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __sv_mns_d(double_t __a, double_t* __b, double_t* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a - __b[i];
  return;
}
/*the minus of double_t in scalar to vector*/
void sv_mns_d(double_t __a, double_t* __b, double_t* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __sv_mns_d<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}
//#####################################################################################################################
__global__ void __vv_mul_b(bool* __a, bool* __b, bool* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a[i] & __b[i];
  return;
}
/*the mul of bool in vector to vector*/
void vv_mul_b(bool* __a, bool* __b, bool* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __vv_mul_b<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __vv_mul_i8(int8_t* __a, int8_t* __b, int8_t* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a[i] * __b[i];
  return;
}
/*the mul of int8_t in vector to vector*/
void vv_mul_i8(int8_t* __a, int8_t* __b, int8_t* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __vv_mul_i8<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __vv_mul_i16(int16_t* __a, int16_t* __b, int16_t* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a[i] * __b[i];
  return;
}
/*the mul of int16_t in vector to vector*/
void vv_mul_i16(int16_t* __a, int16_t* __b, int16_t* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __vv_mul_i16<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __vv_mul_i32(int32_t* __a, int32_t* __b, int32_t* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a[i] * __b[i];
  return;
}
/*the mul of int32_t in vector to vector*/
void vv_mul_i32(int32_t* __a, int32_t* __b, int32_t* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __vv_mul_i32<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __vv_mul_i64(int64_t* __a, int64_t* __b, int64_t* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a[i] * __b[i];
  return;
}
/*the mul of int64_t in vector to vector*/
void vv_mul_i64(int64_t* __a, int64_t* __b, int64_t* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __vv_mul_i64<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __vv_mul_u8(uint8_t* __a, uint8_t* __b, uint8_t* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a[i] * __b[i];
  return;
}
/*the mul of uint8_t in vector to vector*/
void vv_mul_u8(uint8_t* __a, uint8_t* __b, uint8_t* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __vv_mul_u8<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __vv_mul_u16(uint16_t* __a, uint16_t* __b, uint16_t* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a[i] * __b[i];
  return;
}
/*the mul of uint16_t in vector to vector*/
void vv_mul_u16(uint16_t* __a, uint16_t* __b, uint16_t* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __vv_mul_u16<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __vv_mul_u32(uint32_t* __a, uint32_t* __b, uint32_t* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a[i] * __b[i];
  return;
}
/*the mul of uint32_t in vector to vector*/
void vv_mul_u32(uint32_t* __a, uint32_t* __b, uint32_t* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __vv_mul_u32<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __vv_mul_u64(uint64_t* __a, uint64_t* __b, uint64_t* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a[i] * __b[i];
  return;
}
/*the mul of uint64_t in vector to vector*/
void vv_mul_u64(uint64_t* __a, uint64_t* __b, uint64_t* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __vv_mul_u64<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __vv_mul_f(float_t* __a, float_t* __b, float_t* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a[i] * __b[i];
  return;
}
/*the mul of float_t in vector to vector*/
void vv_mul_f(float_t* __a, float_t* __b, float_t* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __vv_mul_f<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __vv_mul_d(double_t* __a, double_t* __b, double_t* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a[i] * __b[i];
  return;
}
/*the mul of double_t in vector to vector*/
void vv_mul_d(double_t* __a, double_t* __b, double_t* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __vv_mul_d<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}
//#####################################################################################################################
__global__ void __vs_mul_b(bool* __a, bool __b, bool* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a[i] & __b;
  return;
}
/*the mul of bool in vector to scalar*/
void vs_mul_b(bool* __a, bool __b, bool* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __vs_mul_b<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __vs_mul_i8(int8_t* __a, int8_t __b, int8_t* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a[i] * __b;
  return;
}
/*the mul of int8_t in vector to scalar*/
void vs_mul_i8(int8_t* __a, int8_t __b, int8_t* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __vs_mul_i8<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __vs_mul_i16(int16_t* __a, int16_t __b, int16_t* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a[i] * __b;
  return;
}
/*the mul of int16_t in vector to scalar*/
void vs_mul_i16(int16_t* __a, int16_t __b, int16_t* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __vs_mul_i16<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __vs_mul_i32(int32_t* __a, int32_t __b, int32_t* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a[i] * __b;
  return;
}
/*the mul of int32_t in vector to scalar*/
void vs_mul_i32(int32_t* __a, int32_t __b, int32_t* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __vs_mul_i32<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __vs_mul_i64(int64_t* __a, int64_t __b, int64_t* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a[i] * __b;
  return;
}
/*the mul of int64_t in vector to scalar*/
void vs_mul_i64(int64_t* __a, int64_t __b, int64_t* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __vs_mul_i64<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __vs_mul_u8(uint8_t* __a, uint8_t __b, uint8_t* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a[i] * __b;
  return;
}
/*the mul of uint8_t in vector to scalar*/
void vs_mul_u8(uint8_t* __a, uint8_t __b, uint8_t* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __vs_mul_u8<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __vs_mul_u16(uint16_t* __a, uint16_t __b, uint16_t* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a[i] * __b;
  return;
}
/*the mul of uint16_t in vector to scalar*/
void vs_mul_u16(uint16_t* __a, uint16_t __b, uint16_t* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __vs_mul_u16<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __vs_mul_u32(uint32_t* __a, uint32_t __b, uint32_t* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a[i] * __b;
  return;
}
/*the mul of uint32_t in vector to scalar*/
void vs_mul_u32(uint32_t* __a, uint32_t __b, uint32_t* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __vs_mul_u32<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __vs_mul_u64(uint64_t* __a, uint64_t __b, uint64_t* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a[i] * __b;
  return;
}
/*the mul of uint64_t in vector to scalar*/
void vs_mul_u64(uint64_t* __a, uint64_t __b, uint64_t* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __vs_mul_u64<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __vs_mul_f(float_t* __a, float_t __b, float_t* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a[i] * __b;
  return;
}
/*the mul of float_t in vector to scalar*/
void vs_mul_f(float_t* __a, float_t __b, float_t* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __vs_mul_f<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __vs_mul_d(double_t* __a, double_t __b, double_t* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a[i] * __b;
  return;
}
/*the mul of double_t in vector to scalar*/
void vs_mul_d(double_t* __a, double_t __b, double_t* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __vs_mul_d<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}
//#####################################################################################################################
__global__ void __vv_div_b(bool* __a, bool* __b, bool* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a[i] ^ __b[i];
  return;
}
/*the divide of bool in vector to vector*/
void vv_div_b(bool* __a, bool* __b, bool* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __vv_div_b<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __vv_div_i8(int8_t* __a, int8_t* __b, int8_t* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a[i] / __b[i];
  return;
}
/*the divide of int8_t in vector to vector*/
void vv_div_i8(int8_t* __a, int8_t* __b, int8_t* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __vv_div_i8<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __vv_div_i16(int16_t* __a, int16_t* __b, int16_t* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a[i] / __b[i];
  return;
}
/*the divide of int16_t in vector to vector*/
void vv_div_i16(int16_t* __a, int16_t* __b, int16_t* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __vv_div_i16<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __vv_div_i32(int32_t* __a, int32_t* __b, int32_t* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a[i] / __b[i];
  return;
}
/*the divide of int32_t in vector to vector*/
void vv_div_i32(int32_t* __a, int32_t* __b, int32_t* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __vv_div_i32<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __vv_div_i64(int64_t* __a, int64_t* __b, int64_t* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a[i] / __b[i];
  return;
}
/*the divide of int64_t in vector to vector*/
void vv_div_i64(int64_t* __a, int64_t* __b, int64_t* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __vv_div_i64<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __vv_div_u8(uint8_t* __a, uint8_t* __b, uint8_t* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a[i] / __b[i];
  return;
}
/*the divide of uint8_t in vector to vector*/
void vv_div_u8(uint8_t* __a, uint8_t* __b, uint8_t* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __vv_div_u8<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __vv_div_u16(uint16_t* __a, uint16_t* __b, uint16_t* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a[i] / __b[i];
  return;
}
/*the divide of uint16_t in vector to vector*/
void vv_div_u16(uint16_t* __a, uint16_t* __b, uint16_t* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __vv_div_u16<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __vv_div_u32(uint32_t* __a, uint32_t* __b, uint32_t* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a[i] / __b[i];
  return;
}
/*the divide of uint32_t in vector to vector*/
void vv_div_u32(uint32_t* __a, uint32_t* __b, uint32_t* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __vv_div_u32<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __vv_div_u64(uint64_t* __a, uint64_t* __b, uint64_t* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a[i] / __b[i];
  return;
}
/*the divide of uint64_t in vector to vector*/
void vv_div_u64(uint64_t* __a, uint64_t* __b, uint64_t* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __vv_div_u64<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __vv_div_f(float_t* __a, float_t* __b, float_t* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a[i] / __b[i];
  return;
}
/*the divide of float_t in vector to vector*/
void vv_div_f(float_t* __a, float_t* __b, float_t* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __vv_div_f<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __vv_div_d(double_t* __a, double_t* __b, double_t* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a[i] / __b[i];
  return;
}
/*the divide of double_t in vector to vector*/
void vv_div_d(double_t* __a, double_t* __b, double_t* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __vv_div_d<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}
//#####################################################################################################################
__global__ void __vs_div_b(bool* __a, bool __b, bool* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a[i] ^ __b;
  return;
}
/*the divide of bool in vector to scalar*/
void vs_div_b(bool* __a, bool __b, bool* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __vs_div_b<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __vs_div_i8(int8_t* __a, int8_t __b, int8_t* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a[i] / __b;
  return;
}
/*the divide of int8_t in vector to scalar*/
void vs_div_i8(int8_t* __a, int8_t __b, int8_t* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __vs_div_i8<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __vs_div_i16(int16_t* __a, int16_t __b, int16_t* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a[i] / __b;
  return;
}
/*the divide of int16_t in vector to scalar*/
void vs_div_i16(int16_t* __a, int16_t __b, int16_t* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __vs_div_i16<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __vs_div_i32(int32_t* __a, int32_t __b, int32_t* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a[i] / __b;
  return;
}
/*the divide of int32_t in vector to scalar*/
void vs_div_i32(int32_t* __a, int32_t __b, int32_t* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __vs_div_i32<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __vs_div_i64(int64_t* __a, int64_t __b, int64_t* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a[i] / __b;
  return;
}
/*the divide of int64_t in vector to scalar*/
void vs_div_i64(int64_t* __a, int64_t __b, int64_t* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __vs_div_i64<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __vs_div_u8(uint8_t* __a, uint8_t __b, uint8_t* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a[i] / __b;
  return;
}
/*the divide of uint8_t in vector to scalar*/
void vs_div_u8(uint8_t* __a, uint8_t __b, uint8_t* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __vs_div_u8<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __vs_div_u16(uint16_t* __a, uint16_t __b, uint16_t* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a[i] / __b;
  return;
}
/*the divide of uint16_t in vector to scalar*/
void vs_div_u16(uint16_t* __a, uint16_t __b, uint16_t* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __vs_div_u16<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __vs_div_u32(uint32_t* __a, uint32_t __b, uint32_t* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a[i] / __b;
  return;
}
/*the divide of uint32_t in vector to scalar*/
void vs_div_u32(uint32_t* __a, uint32_t __b, uint32_t* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __vs_div_u32<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __vs_div_u64(uint64_t* __a, uint64_t __b, uint64_t* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a[i] / __b;
  return;
}
/*the divide of uint64_t in vector to scalar*/
void vs_div_u64(uint64_t* __a, uint64_t __b, uint64_t* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __vs_div_u64<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __vs_div_f(float_t* __a, float_t __b, float_t* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a[i] / __b;
  return;
}
/*the divide of float_t in vector to scalar*/
void vs_div_f(float_t* __a, float_t __b, float_t* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __vs_div_f<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __vs_div_d(double_t* __a, double_t __b, double_t* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a[i] / __b;
  return;
}
/*the divide of double_t in vector to scalar*/
void vs_div_d(double_t* __a, double_t __b, double_t* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __vs_div_d<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}
//#####################################################################################################################
__global__ void __sv_div_b(bool __a, bool* __b, bool* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a ^ __b[i];
  return;
}
/*the divide of bool in scalar to vector*/
void sv_div_b(bool __a, bool* __b, bool* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __sv_div_b<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __sv_div_i8(int8_t __a, int8_t* __b, int8_t* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a / __b[i];
  return;
}
/*the divide of int8_t in scalar to vector*/
void sv_div_i8(int8_t __a, int8_t* __b, int8_t* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __sv_div_i8<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __sv_div_i16(int16_t __a, int16_t* __b, int16_t* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a / __b[i];
  return;
}
/*the divide of int16_t in scalar to vector*/
void sv_div_i16(int16_t __a, int16_t* __b, int16_t* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __sv_div_i16<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __sv_div_i32(int32_t __a, int32_t* __b, int32_t* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a / __b[i];
  return;
}
/*the divide of int32_t in scalar to vector*/
void sv_div_i32(int32_t __a, int32_t* __b, int32_t* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __sv_div_i32<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __sv_div_i64(int64_t __a, int64_t* __b, int64_t* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a / __b[i];
  return;
}
/*the divide of int64_t in scalar to vector*/
void sv_div_i64(int64_t __a, int64_t* __b, int64_t* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __sv_div_i64<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __sv_div_u8(uint8_t __a, uint8_t* __b, uint8_t* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a / __b[i];
  return;
}
/*the divide of uint8_t in scalar to vector*/
void sv_div_u8(uint8_t __a, uint8_t* __b, uint8_t* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __sv_div_u8<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __sv_div_u16(uint16_t __a, uint16_t* __b, uint16_t* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a / __b[i];
  return;
}
/*the divide of uint16_t in scalar to vector*/
void sv_div_u16(uint16_t __a, uint16_t* __b, uint16_t* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __sv_div_u16<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __sv_div_u32(uint32_t __a, uint32_t* __b, uint32_t* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a / __b[i];
  return;
}
/*the divide of uint32_t in scalar to vector*/
void sv_div_u32(uint32_t __a, uint32_t* __b, uint32_t* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __sv_div_u32<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __sv_div_u64(uint64_t __a, uint64_t* __b, uint64_t* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a / __b[i];
  return;
}
/*the divide of uint64_t in scalar to vector*/
void sv_div_u64(uint64_t __a, uint64_t* __b, uint64_t* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __sv_div_u64<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __sv_div_f(float_t __a, float_t* __b, float_t* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a / __b[i];
  return;
}
/*the divide of float_t in scalar to vector*/
void sv_div_f(float_t __a, float_t* __b, float_t* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __sv_div_f<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __sv_div_d(double_t __a, double_t* __b, double_t* __c, size_t __size, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
    __c[i] = __a / __b[i];
  return;
}
/*the divide of double_t in scalar to vector*/
void sv_div_d(double_t __a, double_t* __b, double_t* __c, size_t __size) {
  size_t tuple = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
  size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
  __sv_div_d<<<1, tuples>>>(__a, __b, __c, __size, tuple);
  hipDeviceSynchronize();
  return;
}
//#####################################################################################################################
__global__ void __sum_b(bool* __lib, size_t* __obj, size_t __offest, size_t __strike, size_t __lenth, size_t __strikes, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  bool* ptr;
  for(size_t i = position, j = 0; (i < __lenth) && (j < __tuple); i++, j++){
    ptr = __lib + i * __offest;
    for(size_t j = 0; j < __strikes; j++){
      __obj[i] += (*ptr) ? 1 : 0;
      ptr += __strike;
    }
  }
  return;
}
/*the horizontal summer of bool*/
void h_sum_b(bool* __lib, size_t* __obj, size_t __rows, size_t __cols) {
  size_t tuple = __rows % THREADS ? __rows / THREADS + 1 : __rows / THREADS;
  size_t tuples = __rows % tuple ? __rows / tuple + 1 : __rows / tuple;
  __sum_b<<<1, tuples>>>(__lib, __obj, __cols, 1, __rows, __cols, tuple);
  hipDeviceSynchronize();
  return;
}
/*the vertical summer of bool*/
void v_sum_b(bool* __lib, size_t* __obj, size_t __rows, size_t __cols) {
  size_t tuple = __cols % THREADS ? __cols / THREADS + 1 : __cols / THREADS;
  size_t tuples = __cols % tuple ? __cols / tuple + 1 : __cols / tuple;
  __sum_b<<<1, tuples>>>(__lib, __obj, 1, __cols, __cols, __rows, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __sum_i8(int8_t* __lib, int8_t* __obj, size_t __offest, size_t __strike, size_t __lenth, size_t __strikes, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  int8_t* ptr;
  for(size_t i = position, j = 0; (i < __lenth) && (j < __tuple); i++, j++){
    ptr = __lib + i * __offest;
    for(size_t j = 0; j < __strikes; j++){
      __obj[i] += (*ptr);
      ptr += __strike;
    }
  }
  return;
}
/*the horizontal summer of int8_t*/
void h_sum_i8(int8_t* __lib, int8_t* __obj, size_t __rows, size_t __cols) {
  size_t tuple = __rows % THREADS ? __rows / THREADS + 1 : __rows / THREADS;
  size_t tuples = __rows % tuple ? __rows / tuple + 1 : __rows / tuple;
  __sum_i8<<<1, tuples>>>(__lib, __obj, __cols, 1, __rows, __cols, tuple);
  hipDeviceSynchronize();
  return;
}
/*the vertical summer of int8_t*/
void v_sum_i8(int8_t* __lib, int8_t* __obj, size_t __rows, size_t __cols) {
  size_t tuple = __cols % THREADS ? __cols / THREADS + 1 : __cols / THREADS;
  size_t tuples = __cols % tuple ? __cols / tuple + 1 : __cols / tuple;
  __sum_i8<<<1, tuples>>>(__lib, __obj, 1, __cols, __cols, __rows, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __sum_i16(int16_t* __lib, int16_t* __obj, size_t __offest, size_t __strike, size_t __lenth, size_t __strikes, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  int16_t* ptr;
  for(size_t i = position, j = 0; (i < __lenth) && (j < __tuple); i++, j++){
    ptr = __lib + i * __offest;
    for(size_t j = 0; j < __strikes; j++){
      __obj[i] += (*ptr);
      ptr += __strike;
    }
  }
  return;
}
/*the horizontal summer of int16_t*/
void h_sum_i16(int16_t* __lib, int16_t* __obj, size_t __rows, size_t __cols) {
  size_t tuple = __rows % THREADS ? __rows / THREADS + 1 : __rows / THREADS;
  size_t tuples = __rows % tuple ? __rows / tuple + 1 : __rows / tuple;
  __sum_i16<<<1, tuples>>>(__lib, __obj, __cols, 1, __rows, __cols, tuple);
  hipDeviceSynchronize();
  return;
}
/*the vertical summer of int16_t*/
void v_sum_i16(int16_t* __lib, int16_t* __obj, size_t __rows, size_t __cols) {
  size_t tuple = __cols % THREADS ? __cols / THREADS + 1 : __cols / THREADS;
  size_t tuples = __cols % tuple ? __cols / tuple + 1 : __cols / tuple;
  __sum_i16<<<1, tuples>>>(__lib, __obj, 1, __cols, __cols, __rows, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __sum_i32(int32_t* __lib, int32_t* __obj, size_t __offest, size_t __strike, size_t __lenth, size_t __strikes, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  int32_t* ptr;
  for(size_t i = position, j = 0; (i < __lenth) && (j < __tuple); i++, j++){
    ptr = __lib + i * __offest;
    for(size_t j = 0; j < __strikes; j++){
      __obj[i] += (*ptr);
      ptr += __strike;
    }
  }
  return;
}
/*the horizontal summer of int32_t*/
void h_sum_i32(int32_t* __lib, int32_t* __obj, size_t __rows, size_t __cols) {
  size_t tuple = __rows % THREADS ? __rows / THREADS + 1 : __rows / THREADS;
  size_t tuples = __rows % tuple ? __rows / tuple + 1 : __rows / tuple;
  __sum_i32<<<1, tuples>>>(__lib, __obj, __cols, 1, __rows, __cols, tuple);
  hipDeviceSynchronize();
  return;
}
/*the vertical summer of int32_t*/
void v_sum_i32(int32_t* __lib, int32_t* __obj, size_t __rows, size_t __cols) {
  size_t tuple = __cols % THREADS ? __cols / THREADS + 1 : __cols / THREADS;
  size_t tuples = __cols % tuple ? __cols / tuple + 1 : __cols / tuple;
  __sum_i32<<<1, tuples>>>(__lib, __obj, 1, __cols, __cols, __rows, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __sum_i64(int64_t* __lib, int64_t* __obj, size_t __offest, size_t __strike, size_t __lenth, size_t __strikes, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  int64_t* ptr;
  for(size_t i = position, j = 0; (i < __lenth) && (j < __tuple); i++, j++){
    ptr = __lib + i * __offest;
    for(size_t j = 0; j < __strikes; j++){
      __obj[i] += (*ptr);
      ptr += __strike;
    }
  }
  return;
}
/*the horizontal summer of int64_t*/
void h_sum_i64(int64_t* __lib, int64_t* __obj, size_t __rows, size_t __cols) {
  size_t tuple = __rows % THREADS ? __rows / THREADS + 1 : __rows / THREADS;
  size_t tuples = __rows % tuple ? __rows / tuple + 1 : __rows / tuple;
  __sum_i64<<<1, tuples>>>(__lib, __obj, __cols, 1, __rows, __cols, tuple);
  hipDeviceSynchronize();
  return;
}
/*the vertical summer of int64_t*/
void v_sum_i64(int64_t* __lib, int64_t* __obj, size_t __rows, size_t __cols) {
  size_t tuple = __cols % THREADS ? __cols / THREADS + 1 : __cols / THREADS;
  size_t tuples = __cols % tuple ? __cols / tuple + 1 : __cols / tuple;
  __sum_i64<<<1, tuples>>>(__lib, __obj, 1, __cols, __cols, __rows, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __sum_u8(uint8_t* __lib, uint8_t* __obj, size_t __offest, size_t __strike, size_t __lenth, size_t __strikes, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  uint8_t* ptr;
  for(size_t i = position, j = 0; (i < __lenth) && (j < __tuple); i++, j++){
    ptr = __lib + i * __offest;
    for(size_t j = 0; j < __strikes; j++){
      __obj[i] += (*ptr);
      ptr += __strike;
    }
  }
  return;
}
/*the horizontal summer of uint8_t*/
void h_sum_u8(uint8_t* __lib, uint8_t* __obj, size_t __rows, size_t __cols) {
  size_t tuple = __rows % THREADS ? __rows / THREADS + 1 : __rows / THREADS;
  size_t tuples = __rows % tuple ? __rows / tuple + 1 : __rows / tuple;
  __sum_u8<<<1, tuples>>>(__lib, __obj, __cols, 1, __rows, __cols, tuple);
  hipDeviceSynchronize();
  return;
}
/*the vertical summer of uint8_t*/
void v_sum_u8(uint8_t* __lib, uint8_t* __obj, size_t __rows, size_t __cols) {
  size_t tuple = __cols % THREADS ? __cols / THREADS + 1 : __cols / THREADS;
  size_t tuples = __cols % tuple ? __cols / tuple + 1 : __cols / tuple;
  __sum_u8<<<1, tuples>>>(__lib, __obj, 1, __cols, __cols, __rows, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __sum_u16(uint16_t* __lib, uint16_t* __obj, size_t __offest, size_t __strike, size_t __lenth, size_t __strikes, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  uint16_t* ptr;
  for(size_t i = position, j = 0; (i < __lenth) && (j < __tuple); i++, j++){
    ptr = __lib + i * __offest;
    for(size_t j = 0; j < __strikes; j++){
      __obj[i] += (*ptr);
      ptr += __strike;
    }
  }
  return;
}
/*the horizontal summer of uint16_t*/
void h_sum_u16(uint16_t* __lib, uint16_t* __obj, size_t __rows, size_t __cols) {
  size_t tuple = __rows % THREADS ? __rows / THREADS + 1 : __rows / THREADS;
  size_t tuples = __rows % tuple ? __rows / tuple + 1 : __rows / tuple;
  __sum_u16<<<1, tuples>>>(__lib, __obj, __cols, 1, __rows, __cols, tuple);
  hipDeviceSynchronize();
  return;
}
/*the vertical summer of uint16_t*/
void v_sum_u16(uint16_t* __lib, uint16_t* __obj, size_t __rows, size_t __cols) {
  size_t tuple = __cols % THREADS ? __cols / THREADS + 1 : __cols / THREADS;
  size_t tuples = __cols % tuple ? __cols / tuple + 1 : __cols / tuple;
  __sum_u16<<<1, tuples>>>(__lib, __obj, 1, __cols, __cols, __rows, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __sum_u32(uint32_t* __lib, uint32_t* __obj, size_t __offest, size_t __strike, size_t __lenth, size_t __strikes, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  uint32_t* ptr;
  for(size_t i = position, j = 0; (i < __lenth) && (j < __tuple); i++, j++){
    ptr = __lib + i * __offest;
    for(size_t j = 0; j < __strikes; j++){
      __obj[i] += (*ptr);
      ptr += __strike;
    }
  }
  return;
}
/*the horizontal summer of uint32_t*/
void h_sum_u32(uint32_t* __lib, uint32_t* __obj, size_t __rows, size_t __cols) {
  size_t tuple = __rows % THREADS ? __rows / THREADS + 1 : __rows / THREADS;
  size_t tuples = __rows % tuple ? __rows / tuple + 1 : __rows / tuple;
  __sum_u32<<<1, tuples>>>(__lib, __obj, __cols, 1, __rows, __cols, tuple);
  hipDeviceSynchronize();
  return;
}
/*the vertical summer of uint32_t*/
void v_sum_u32(uint32_t* __lib, uint32_t* __obj, size_t __rows, size_t __cols) {
  size_t tuple = __cols % THREADS ? __cols / THREADS + 1 : __cols / THREADS;
  size_t tuples = __cols % tuple ? __cols / tuple + 1 : __cols / tuple;
  __sum_u32<<<1, tuples>>>(__lib, __obj, 1, __cols, __cols, __rows, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __sum_u64(uint64_t* __lib, uint64_t* __obj, size_t __offest, size_t __strike, size_t __lenth, size_t __strikes, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  uint64_t* ptr;
  for(size_t i = position, j = 0; (i < __lenth) && (j < __tuple); i++, j++){
    ptr = __lib + i * __offest;
    for(size_t j = 0; j < __strikes; j++){
      __obj[i] += (*ptr);
      ptr += __strike;
    }
  }
  return;
}
/*the horizontal summer of uint64_t*/
void h_sum_u64(uint64_t* __lib, uint64_t* __obj, size_t __rows, size_t __cols) {
  size_t tuple = __rows % THREADS ? __rows / THREADS + 1 : __rows / THREADS;
  size_t tuples = __rows % tuple ? __rows / tuple + 1 : __rows / tuple;
  __sum_u64<<<1, tuples>>>(__lib, __obj, __cols, 1, __rows, __cols, tuple);
  hipDeviceSynchronize();
  return;
}
/*the vertical summer of uint64_t*/
void v_sum_u64(uint64_t* __lib, uint64_t* __obj, size_t __rows, size_t __cols) {
  size_t tuple = __cols % THREADS ? __cols / THREADS + 1 : __cols / THREADS;
  size_t tuples = __cols % tuple ? __cols / tuple + 1 : __cols / tuple;
  __sum_u64<<<1, tuples>>>(__lib, __obj, 1, __cols, __cols, __rows, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __sum_f(float_t* __lib, float_t* __obj, size_t __offest, size_t __strike, size_t __lenth, size_t __strikes, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  float_t* ptr;
  for(size_t i = position, j = 0; (i < __lenth) && (j < __tuple); i++, j++){
    ptr = __lib + i * __offest;
    for(size_t j = 0; j < __strikes; j++){
      __obj[i] += (*ptr);
      ptr += __strike;
    }
  }
  return;
}
/*the horizontal summer of float_t*/
void h_sum_f(float_t* __lib, float_t* __obj, size_t __rows, size_t __cols) {
  size_t tuple = __rows % THREADS ? __rows / THREADS + 1 : __rows / THREADS;
  size_t tuples = __rows % tuple ? __rows / tuple + 1 : __rows / tuple;
  __sum_f<<<1, tuples>>>(__lib, __obj, __cols, 1, __rows, __cols, tuple);
  hipDeviceSynchronize();
  return;
}
/*the vertical summer of float_t*/
void v_sum_f(float_t* __lib, float_t* __obj, size_t __rows, size_t __cols) {
  size_t tuple = __cols % THREADS ? __cols / THREADS + 1 : __cols / THREADS;
  size_t tuples = __cols % tuple ? __cols / tuple + 1 : __cols / tuple;
  __sum_f<<<1, tuples>>>(__lib, __obj, 1, __cols, __cols, __rows, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __sum_d(double_t* __lib, double_t* __obj, size_t __offest, size_t __strike, size_t __lenth, size_t __strikes, size_t __tuple) {
  size_t position = threadIdx.x;
  position *= __tuple;
  double_t* ptr;
  for(size_t i = position, j = 0; (i < __lenth) && (j < __tuple); i++, j++){
    ptr = __lib + i * __offest;
    for(size_t j = 0; j < __strikes; j++){
      __obj[i] += (*ptr);
      ptr += __strike;
    }
  }
  return;
}
/*the horizontal summer of double_t*/
void h_sum_d(double_t* __lib, double_t* __obj, size_t __rows, size_t __cols) {
  size_t tuple = __rows % THREADS ? __rows / THREADS + 1 : __rows / THREADS;
  size_t tuples = __rows % tuple ? __rows / tuple + 1 : __rows / tuple;
  __sum_d<<<1, tuples>>>(__lib, __obj, __cols, 1, __rows, __cols, tuple);
  hipDeviceSynchronize();
  return;
}
/*the vertical summer of double_t*/
void v_sum_d(double_t* __lib, double_t* __obj, size_t __rows, size_t __cols) {
  size_t tuple = __cols % THREADS ? __cols / THREADS + 1 : __cols / THREADS;
  size_t tuples = __cols % tuple ? __cols / tuple + 1 : __cols / tuple;
  __sum_d<<<1, tuples>>>(__lib, __obj, 1, __cols, __cols, __rows, tuple);
  hipDeviceSynchronize();
  return;
}
//#####################################################################################################################
__global__ void __dot_b(bool* __a, bool* __b, bool* __c, size_t __row, size_t __line, size_t __col, size_t __tuple) {
  size_t position = threadIdx.x, total = __row * __col;
  position *= __tuple;
  size_t pR, pC;
  auto a = __a, b = __b;
  for(size_t i = position, j = 0; (i < total) && (j < __tuple); i++, j++){
    pR = i / __col;
    pC = i % __col;
    a = __a + __line * pR;
    b = __b + pC;
    for(size_t w = 0; w < __line; w++){
      __c[i] |= (*a) & (*b);
      a++;
      b += __col;
    }
  }
}
/*dot of bool*/
void dot_b(bool* __a, bool* __b, bool* __c, size_t __row, size_t __line, size_t __col){
  size_t total = __row * __col;
  size_t tuple = total % THREADS ? total / THREADS + 1 : total / THREADS;
  size_t tuples = total % tuple ? total / tuple + 1 : total / tuple;
  __dot_b<<<1, tuples>>>(__a, __b, __c, __row, __line, __col, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __dot_i8(int8_t* __a, int8_t* __b, int8_t* __c, size_t __row, size_t __line, size_t __col, size_t __tuple) {
  size_t position = threadIdx.x, total = __row * __col;
  position *= __tuple;
  size_t pR, pC;
  auto a = __a, b = __b;
  for(size_t i = position, j = 0; (i < total) && (j < __tuple); i++, j++){
    pR = i / __col;
    pC = i % __col;
    a = __a + __line * pR;
    b = __b + pC;
    for(size_t w = 0; w < __line; w++){
      __c[i] += (*a) * (*b);
      a++;
      b += __col;
    }
  }
}
/*dot of int8_t*/
void dot_i8(int8_t* __a, int8_t* __b, int8_t* __c, size_t __row, size_t __line, size_t __col){
  size_t total = __row * __col;
  size_t tuple = total % THREADS ? total / THREADS + 1 : total / THREADS;
  size_t tuples = total % tuple ? total / tuple + 1 : total / tuple;
  __dot_i8<<<1, tuples>>>(__a, __b, __c, __row, __line, __col, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __dot_i16(int16_t* __a, int16_t* __b, int16_t* __c, size_t __row, size_t __line, size_t __col, size_t __tuple) {
  size_t position = threadIdx.x, total = __row * __col;
  position *= __tuple;
  size_t pR, pC;
  auto a = __a, b = __b;
  for(size_t i = position, j = 0; (i < total) && (j < __tuple); i++, j++){
    pR = i / __col;
    pC = i % __col;
    a = __a + __line * pR;
    b = __b + pC;
    for(size_t w = 0; w < __line; w++){
      __c[i] += (*a) * (*b);
      a++;
      b += __col;
    }
  }
}
/*dot of int16_t*/
void dot_i16(int16_t* __a, int16_t* __b, int16_t* __c, size_t __row, size_t __line, size_t __col){
  size_t total = __row * __col;
  size_t tuple = total % THREADS ? total / THREADS + 1 : total / THREADS;
  size_t tuples = total % tuple ? total / tuple + 1 : total / tuple;
  __dot_i16<<<1, tuples>>>(__a, __b, __c, __row, __line, __col, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __dot_i32(int32_t* __a, int32_t* __b, int32_t* __c, size_t __row, size_t __line, size_t __col, size_t __tuple) {
  size_t position = threadIdx.x, total = __row * __col;
  position *= __tuple;
  size_t pR, pC;
  auto a = __a, b = __b;
  for(size_t i = position, j = 0; (i < total) && (j < __tuple); i++, j++){
    pR = i / __col;
    pC = i % __col;
    a = __a + __line * pR;
    b = __b + pC;
    for(size_t w = 0; w < __line; w++){
      __c[i] += (*a) * (*b);
      a++;
      b += __col;
    }
  }
}
/*dot of int32_t*/
void dot_i32(int32_t* __a, int32_t* __b, int32_t* __c, size_t __row, size_t __line, size_t __col){
  size_t total = __row * __col;
  size_t tuple = total % THREADS ? total / THREADS + 1 : total / THREADS;
  size_t tuples = total % tuple ? total / tuple + 1 : total / tuple;
  __dot_i32<<<1, tuples>>>(__a, __b, __c, __row, __line, __col, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __dot_i64(int64_t* __a, int64_t* __b, int64_t* __c, size_t __row, size_t __line, size_t __col, size_t __tuple) {
  size_t position = threadIdx.x, total = __row * __col;
  position *= __tuple;
  size_t pR, pC;
  auto a = __a, b = __b;
  for(size_t i = position, j = 0; (i < total) && (j < __tuple); i++, j++){
    pR = i / __col;
    pC = i % __col;
    a = __a + __line * pR;
    b = __b + pC;
    for(size_t w = 0; w < __line; w++){
      __c[i] += (*a) * (*b);
      a++;
      b += __col;
    }
  }
}
/*dot of int64_t*/
void dot_i64(int64_t* __a, int64_t* __b, int64_t* __c, size_t __row, size_t __line, size_t __col){
  size_t total = __row * __col;
  size_t tuple = total % THREADS ? total / THREADS + 1 : total / THREADS;
  size_t tuples = total % tuple ? total / tuple + 1 : total / tuple;
  __dot_i64<<<1, tuples>>>(__a, __b, __c, __row, __line, __col, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __dot_u8(uint8_t* __a, uint8_t* __b, uint8_t* __c, size_t __row, size_t __line, size_t __col, size_t __tuple) {
  size_t position = threadIdx.x, total = __row * __col;
  position *= __tuple;
  size_t pR, pC;
  auto a = __a, b = __b;
  for(size_t i = position, j = 0; (i < total) && (j < __tuple); i++, j++){
    pR = i / __col;
    pC = i % __col;
    a = __a + __line * pR;
    b = __b + pC;
    for(size_t w = 0; w < __line; w++){
      __c[i] += (*a) * (*b);
      a++;
      b += __col;
    }
  }
}
/*dot of uint8_t*/
void dot_u8(uint8_t* __a, uint8_t* __b, uint8_t* __c, size_t __row, size_t __line, size_t __col){
  size_t total = __row * __col;
  size_t tuple = total % THREADS ? total / THREADS + 1 : total / THREADS;
  size_t tuples = total % tuple ? total / tuple + 1 : total / tuple;
  __dot_u8<<<1, tuples>>>(__a, __b, __c, __row, __line, __col, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __dot_u16(uint16_t* __a, uint16_t* __b, uint16_t* __c, size_t __row, size_t __line, size_t __col, size_t __tuple) {
  size_t position = threadIdx.x, total = __row * __col;
  position *= __tuple;
  size_t pR, pC;
  auto a = __a, b = __b;
  for(size_t i = position, j = 0; (i < total) && (j < __tuple); i++, j++){
    pR = i / __col;
    pC = i % __col;
    a = __a + __line * pR;
    b = __b + pC;
    for(size_t w = 0; w < __line; w++){
      __c[i] += (*a) * (*b);
      a++;
      b += __col;
    }
  }
}
/*dot of uint16_t*/
void dot_u16(uint16_t* __a, uint16_t* __b, uint16_t* __c, size_t __row, size_t __line, size_t __col){
  size_t total = __row * __col;
  size_t tuple = total % THREADS ? total / THREADS + 1 : total / THREADS;
  size_t tuples = total % tuple ? total / tuple + 1 : total / tuple;
  __dot_u16<<<1, tuples>>>(__a, __b, __c, __row, __line, __col, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __dot_u32(uint32_t* __a, uint32_t* __b, uint32_t* __c, size_t __row, size_t __line, size_t __col, size_t __tuple) {
  size_t position = threadIdx.x, total = __row * __col;
  position *= __tuple;
  size_t pR, pC;
  auto a = __a, b = __b;
  for(size_t i = position, j = 0; (i < total) && (j < __tuple); i++, j++){
    pR = i / __col;
    pC = i % __col;
    a = __a + __line * pR;
    b = __b + pC;
    for(size_t w = 0; w < __line; w++){
      __c[i] += (*a) * (*b);
      a++;
      b += __col;
    }
  }
}
/*dot of uint32_t*/
void dot_u32(uint32_t* __a, uint32_t* __b, uint32_t* __c, size_t __row, size_t __line, size_t __col){
  size_t total = __row * __col;
  size_t tuple = total % THREADS ? total / THREADS + 1 : total / THREADS;
  size_t tuples = total % tuple ? total / tuple + 1 : total / tuple;
  __dot_u32<<<1, tuples>>>(__a, __b, __c, __row, __line, __col, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __dot_u64(uint64_t* __a, uint64_t* __b, uint64_t* __c, size_t __row, size_t __line, size_t __col, size_t __tuple) {
  size_t position = threadIdx.x, total = __row * __col;
  position *= __tuple;
  size_t pR, pC;
  auto a = __a, b = __b;
  for(size_t i = position, j = 0; (i < total) && (j < __tuple); i++, j++){
    pR = i / __col;
    pC = i % __col;
    a = __a + __line * pR;
    b = __b + pC;
    for(size_t w = 0; w < __line; w++){
      __c[i] += (*a) * (*b);
      a++;
      b += __col;
    }
  }
}
/*dot of uint64_t*/
void dot_u64(uint64_t* __a, uint64_t* __b, uint64_t* __c, size_t __row, size_t __line, size_t __col){
  size_t total = __row * __col;
  size_t tuple = total % THREADS ? total / THREADS + 1 : total / THREADS;
  size_t tuples = total % tuple ? total / tuple + 1 : total / tuple;
  __dot_u64<<<1, tuples>>>(__a, __b, __c, __row, __line, __col, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __dot_f(float_t* __a, float_t* __b, float_t* __c, size_t __row, size_t __line, size_t __col, size_t __tuple) {
  size_t position = threadIdx.x, total = __row * __col;
  position *= __tuple;
  size_t pR, pC;
  auto a = __a, b = __b;
  for(size_t i = position, j = 0; (i < total) && (j < __tuple); i++, j++){
    pR = i / __col;
    pC = i % __col;
    a = __a + __line * pR;
    b = __b + pC;
    for(size_t w = 0; w < __line; w++){
      __c[i] += (*a) * (*b);
      a++;
      b += __col;
    }
  }
}
/*dot of float_t*/
void dot_f(float_t* __a, float_t* __b, float_t* __c, size_t __row, size_t __line, size_t __col){
  size_t total = __row * __col;
  size_t tuple = total % THREADS ? total / THREADS + 1 : total / THREADS;
  size_t tuples = total % tuple ? total / tuple + 1 : total / tuple;
  __dot_f<<<1, tuples>>>(__a, __b, __c, __row, __line, __col, tuple);
  hipDeviceSynchronize();
  return;
}

__global__ void __dot_d(double_t* __a, double_t* __b, double_t* __c, size_t __row, size_t __line, size_t __col, size_t __tuple) {
  size_t position = threadIdx.x, total = __row * __col;
  position *= __tuple;
  size_t pR, pC;
  auto a = __a, b = __b;
  for(size_t i = position, j = 0; (i < total) && (j < __tuple); i++, j++){
    pR = i / __col;
    pC = i % __col;
    a = __a + __line * pR;
    b = __b + pC;
    for(size_t w = 0; w < __line; w++){
      __c[i] += (*a) * (*b);
      a++;
      b += __col;
    }
  }
}
/*dot of double_t*/
void dot_d(double_t* __a, double_t* __b, double_t* __c, size_t __row, size_t __line, size_t __col){
  size_t total = __row * __col;
  size_t tuple = total % THREADS ? total / THREADS + 1 : total / THREADS;
  size_t tuples = total % tuple ? total / tuple + 1 : total / tuple;
  __dot_d<<<1, tuples>>>(__a, __b, __c, __row, __line, __col, tuple);
  hipDeviceSynchronize();
  return;
}
//#####################################################################################################################

clock_t test_for_vram(float_t* __a, float_t* __b, float_t* __c, size_t __size) {
  float_t *deviceA, *deviceB, *deviceC;
  auto size = __size * sizeof(float_t);
  clock_t first, second;
  hipMalloc(&deviceA, size);
  hipMalloc(&deviceB, size);
  hipMalloc(&deviceC, size);
  hipMemcpy(deviceA, __a, size, hipMemcpyHostToDevice);
  hipMemcpy(deviceB, __b, size, hipMemcpyHostToDevice);
  first = clock();
  vv_add_f(deviceA, deviceB, deviceC, __size);
  second = clock();
  hipMemcpy(__c, deviceC, size, hipMemcpyDeviceToHost);
  hipFree(deviceA);
  hipFree(deviceB);
  hipFree(deviceC);
  return second - first;
}

#ifdef __cplusplus
}
#endif