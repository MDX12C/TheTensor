#include "hip/hip_runtime.h"
#include "thetensor/define.h"

#ifdef __cplusplus
extern "C"{
#endif

__global__ void __vv_add_i32(int32_t* __a, int32_t* __b, int32_t* __c, size_t __size, size_t __tuple = 1) {
  size_t position = threadIdx.x + blockDim.x * blockIdx.x;
  position *= __tuple;
  if (__tuple == 1)
    for(size_t i = position; i < __size; i++)
      __c[i] = __a[i] + __b[i];
  else
    for(size_t i = position, j = 0; (i < __size) && (j < __tuple); i++, j++)
      __c[i] = __a[i] + __b[i];
  return;
}
inline void vv_add_i32(int32_t* __a, int32_t* __b, int32_t* __c, size_t __size) {
  if (__size > GROUP) {
    size_t tuple = __size % GROUP ? __size / GROUP + 1 : __size / GROUP;
    size_t tuples = __size % tuple ? __size / tuple + 1 : __size / tuple;
    size_t blocks = tuples % THREADS ? tuples / THREADS + 1 : tuples / THREADS;
    __vv_add_i32<<<blocks, THREADS>>>(__a, __b, __c, __size, tuple);
  } else if (__size > THREADS) {
    size_t blocks = __size % THREADS ? __size / THREADS + 1 : __size / THREADS;
    __vv_add_i32<<<blocks, THREADS>>>(__a, __b, __c, __size);
  } else {
    __vv_add_i32<<<1, __size>>>(__a, __b, __c, __size);
  }
  return;
}

#ifdef __cplusplus
}
#endif