#include <hip/hip_runtime.h>


#include <stdio.h>

__global__ void kernelFunction() { printf("the cuda kernel from %d\n",threadIdx.x); }
signed main() {
  printf("hello world\n");
  kernelFunction<<<10, 512>>>();
  hipDeviceSynchronize();
  return 0;
}