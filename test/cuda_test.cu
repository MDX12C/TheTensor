#include <hip/hip_runtime.h>


#include <stdio.h>
#include <float.h>
#include <stdint.h>

typedef int Type;
__global__ void mul(Type* __alpha,Type* __beta,Type* __gamma,size_t __lenth){
  auto i=blockDim.x*blockIdx.x+threadIdx.x;
  if (i<__lenth)__gamma[i]=__beta[i]*__alpha[i];
}
signed main() {
  size_t const LENTH=1048576;
  size_t const THREADS=1024;
  size_t const BLOCKS=LENTH/THREADS;
  Type* alpha=(Type*)malloc(LENTH*sizeof(Type));
  Type* beta=(Type*)malloc(LENTH*sizeof(Type));
  Type* gamma=(Type*)malloc(LENTH*sizeof(Type));
  for(size_t i=0;i<LENTH;i++)alpha[i]=beta[i]=16;
  mul<<<BLOCKS,THREADS>>>(alpha,beta,gamma,LENTH);
  hipDeviceSynchronize();
  for(size_t i=0;i<LENTH;i++){
    if (gamma[i]!=(alpha[i]*beta[i])){
      printf("Error operate when i=%ld\n",i);
      break;
    }
  }
  printf("End test");
  free(alpha);
  free(beta);
  free(gamma);
  return 0;
}